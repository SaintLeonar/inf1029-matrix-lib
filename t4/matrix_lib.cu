#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "matrix_lib.h"

#define THREADS_PER_BLOCK 256

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

    return 1;
}

__global__
void kernel_scalar_matrix_mult(float scalar_value, Matrix matrix, unsigned long int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int local = blockDim.x * gridDim.x;

    for (unsigned long int i = index; i < n; i += local) {
	    matrix.d_rows[i] = matrix.d_rows[i] * scalar_value;
    }
}


int scalar_matrix_mult(float scalar_value, Matrix *matrix){
    // declaração de variaveis
    hipError_t hipError_t;
    int blockSize, numBlocks;
    unsigned long int tam;
    tam = matrix->height * matrix->width;

    // checa a matriz
    if (tam == 0 || matrix->d_rows == NULL || matrix->h_rows == NULL) return 0;
    
    // Copia a matrix de host para device
    hipError_t = hipMemcpy(matrix->d_rows, matrix->h_rows, tam*sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess) {
	printf("hipMemcpy (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    // Inicia o Kernel
    blockSize = THREADS_PER_BLOCK;
    numBlocks = (tam + blockSize - 1) / blockSize;
    kernel_scalar_matrix_mult<<<numBlocks, blockSize>>>(scalar_value, *matrix, tam);

    // Espera a GPU terminar
    hipDeviceSynchronize();

    // Copia a matrix de device para host
    hipError_t = hipMemcpy(matrix->h_rows, matrix->d_rows, tam*sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess){
        printf("hipMemcpy (device to host): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
        }

  return 1;
}

__global__
void kernel_matrix_matrix_mult(Matrix matrixA, Matrix matrixB, Matrix matrixC){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int local = blockIdx.x * blockDim.x;

    unsigned long int tamC;
    tamC = matrixC.height * matrixC.width;

    for(int i = index; i < tam; i+= local){
        a = i / matrixC.width;
        b = i % matrixC.width;

        matrixC.d_rows[i] = 0;

        for(int j = 0; j < matrixA.width; j++)
            matrixC.d_rows[i] += matrixA.d_rows[a * matrixA.width + j] * matrixB.d_rows[j * matrixB.height + b];
    }
}


int matrix_matrix_mult(Matrix *matrixA, Matrix * matrixB, Matrix * matrixC){

    int blockSize, numBlocks;
    hipError_t hipError_t;

    unsigned long int tamA, tamB, tamC;

    tamA = matrixA->width * matrixA->height;
    if(tamA == 0 || matrixA->d_rows == NULL || matrixA->h_rows == NULL) return 0;
    tamB = matrixB->width * matrixB->height;
    if(tamB == 0 || matrixB->d_rows == NULL || matrixB->h_rows == NULL) return 0;
    tamC = matrixC->width * matrixC->height;
    if(tamC == 0 || matrixC->d_rows == NULL || matrixC->h_rows == NULL) return 0;

    // Copia a matrix de host para device
    hipError_t = hipMemcpy(matrixA->d_rows, matrixA->h_rows, tamA*sizeof(float), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
	printf("hipMemcpy - A (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    hipError_t = hipMemcpy(matrixB->d_rows, matrixB->h_rows, tamB*sizeof(float), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
	printf("hipMemcpy - B (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

   blockSize = THREADS_PER_BLOCK;
   numBlocks = (tamC + blockSize - 1) / blockSize;
   kernel_matrix_matrix_mult<<<numBlocks, blockSize>>>(*matrixA, *matrixB, *matrixC);

    hipDeviceSynchronize();

    hipError_t = hipMemcpy(matrixC->h_rows, matrixC->d_rows, tamC*sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy - C (device to host): returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 1;
    }

  return 1;
}