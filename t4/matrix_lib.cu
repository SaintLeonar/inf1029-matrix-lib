#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "matrix_lib.h"

#define THREADS_PER_BLOCK 256

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

}

__global__
void kernel_scalar_matrix_mult(float scalar_value, Matrix matrix, unsigned long int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int local = blockDim.x * gridDim.x;

    for (unsigned long int i = index; i < n; i += local) {
	    matrix.d_rows[i] = matrix.d_rows[i] * scalar_value;
    }
}


int scalar_matrix_mult(float scalar_value, Matrix *matrix){
    // declaração de variaveis
    hipError_t hipError_t;
    int blockSize, numBlocks;
    unsigned long int tam;
    tam = matrix->height * matrix->width;

    // checa a matriz
    if (tam == 0 || matrix->d_rows == NULL || matrix->h_rows == NULL) return 0;
    
    // Copia a matrix de host para device
    hipError_t = hipMemcpy(matrix->d_rows, matrix->h_rows, tam*sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess) {
	printf("hipMemcpy (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    // Inicia o Kernel
    blockSize = THREADS_PER_BLOCK;
    numBlocks = (N + blockSize - 1) / blockSize;
    kernel_scalar_matrix_mult<<<numBlocks, blockSize>>>(scalar_value, *matrix, N);

    // Espera a GPU terminar
    hipDeviceSynchronize();

    // Copia a matrix de device para host
    hipError_t = hipMemcpy(matrix->h_rows, matrix->d_rows, N*sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess){
        printf("hipMemcpy (device to host): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
        }

  return 1;
}