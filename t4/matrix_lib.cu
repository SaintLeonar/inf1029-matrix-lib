#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "matrix_lib.h"

int blockSize = 256;
int numBlocks = 4096;

int set_grid_size(int threads_per_block, int max_blocks_per_grid){
    blockSize = threads_per_block;
    numBlocks = max_blocks_per_grid;
    return 1;
}

__global__
void kernel_scalar_matrix_mult(float scalar_value, Matrix matrix, unsigned long int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int local = blockDim.x * gridDim.x;

    for (unsigned long int i = index; i < n; i += local) {
	    matrix.d_rows[i] = matrix.d_rows[i] * scalar_value;
    }
}

__global__ void kernel(Matrix matrixA, Matrix matrixB, Matrix matrixC){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int local = blockDim.x * gridDim.x;
    int posicao;

    unsigned long int tamC, i, j, k;

    tamC = matrixC.height * matrixC.width;

    for (posicao = index; posicao <  tamC; posicao += local) {
    	i = posicao / matrixC.width;
    	j = posicao % matrixC.width;

	matrixC.d_rows[posicao] = 0;

	for (k = 0; k < matrixA.width; ++k) 
		matrixC.d_rows[posicao] += matrixA.d_rows[(i * matrixA.width) + k] * matrixB.d_rows[(k * matrixB.height) + j];
    }
}

int scalar_matrix_mult(float scalar_value, Matrix *matrix){
    // declaração de variaveis
    hipError_t hipError_t;
    unsigned long int tam;
    tam = matrix->height * matrix->width;

    // checa a matriz
    if (tam == 0 || matrix->d_rows == NULL || matrix->h_rows == NULL) return 0;
    
    // Copia a matrix de host para device
    hipError_t = hipMemcpy(matrix->d_rows, matrix->h_rows, tam*sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess) {
	printf("hipMemcpy (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    kernel_scalar_matrix_mult<<<numBlocks, blockSize>>>(scalar_value, *matrix, tam);

    // Espera a GPU terminar
    hipDeviceSynchronize();

    // Copia a matrix de device para host
    hipError_t = hipMemcpy(matrix->h_rows, matrix->d_rows, tam*sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess){
        printf("hipMemcpy (device to host): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
        }

  return 1;
}

int matrix_matrix_mult(Matrix *matrixA, Matrix * matrixB, Matrix * matrixC){
    hipError_t hipError_t;
    unsigned long int tamA, tamB, tamC;

    tamA = matrixA->width * matrixA->height;
    if(tamA == 0 || matrixA->d_rows == NULL || matrixA->h_rows == NULL) return 0;
    tamB = matrixB->width * matrixB->height;
    if(tamB == 0 || matrixB->d_rows == NULL || matrixB->h_rows == NULL) return 0;
    tamC = matrixC->width * matrixC->height;
    if(tamC == 0 || matrixC->d_rows == NULL || matrixC->h_rows == NULL) return 0;

    // Copia a matrix de host para device
    hipError_t = hipMemcpy(matrixA->d_rows, matrixA->h_rows, tamA*sizeof(float), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
	printf("hipMemcpy - A (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    hipError_t = hipMemcpy(matrixB->d_rows, matrixB->h_rows, tamB*sizeof(float), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
	printf("hipMemcpy - B (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }
    
    kernel<<<numBlocks, blockSize>>>(*matrixA, *matrixB, *matrixC);
    
    
    hipDeviceSynchronize();
    
    hipError_t = hipMemcpy(matrixC->h_rows, matrixC->d_rows, tamC*sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy - C (device to host): returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

  return 1;
}