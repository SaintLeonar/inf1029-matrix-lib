#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "matrix_lib.h"

#define THREADS_PER_BLOCK 256

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

    return 1;
}

__global__
void kernel_scalar_matrix_mult(float scalar_value, Matrix matrix, unsigned long int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int local = blockDim.x * gridDim.x;

    for (unsigned long int i = index; i < n; i += local) {
	    matrix.d_rows[i] = matrix.d_rows[i] * scalar_value;
    }
}


int scalar_matrix_mult(float scalar_value, Matrix *matrix){
    // declaração de variaveis
    hipError_t hipError_t;
    int blockSize, numBlocks;
    unsigned long int tam;
    tam = matrix->height * matrix->width;

    // checa a matriz
    if (tam == 0 || matrix->d_rows == NULL || matrix->h_rows == NULL) return 0;
    
    // Copia a matrix de host para device
    hipError_t = hipMemcpy(matrix->d_rows, matrix->h_rows, tam*sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess) {
	printf("hipMemcpy (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    // Inicia o Kernel
    blockSize = THREADS_PER_BLOCK;
    numBlocks = (tam + blockSize - 1) / blockSize;
    kernel_scalar_matrix_mult<<<numBlocks, blockSize>>>(scalar_value, *matrix, tam);

    // Espera a GPU terminar
    hipDeviceSynchronize();

    // Copia a matrix de device para host
    hipError_t = hipMemcpy(matrix->h_rows, matrix->d_rows, tam*sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess){
        printf("hipMemcpy (device to host): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
        }

  return 1;
}

__global__
void kernel_matrix_matrix_mult(Matrix matrixA, Matrix matrixB, Matrix matrixC){
    int largura = blockIdx.x * blockDim.x + threadIdx.x;
    int altura = blockIdx.y * blockDim.y + threadIdx.y;

    float soma = 0;

    if(largura < THREADS_PER_BLOCK && altura < THREADS_PER_BLOCK)
        for(int i = 0; i < THREADS_PER_BLOCK; i++)
            matrixC.d_rows[largura * THREADS_PER_BLOCK + altura] += matrixA.d_rows[largura * THREADS_PER_BLOCK + i] * matrixB.d_rows[i * THREADS_PER_BLOCK + altura];
}


int matrix_matrix_mult(Matrix *matrixA, Matrix * matrixB, Matrix * matrixC){

    int blockSize, numBlocks;
    hipError_t hipError_t;

    unsigned long int tamA, tamB, tamC;

    tamA = matrixA->width * matrixA->height;
    if(tamA == 0 || matrixA->d_rows == NULL || matrixA->h_rows == NULL) return 0;
    tamB = matrixB->width * matrixB->height;
    if(tamB == 0 || matrixB->d_rows == NULL || matrixB->h_rows == NULL) return 0;
    tamC = matrixC->width * matrixC->height;
    if(tamC == 0 || matrixC->d_rows == NULL || matrixC->h_rows == NULL) return 0;

    // Copia a matrix de host para device
    hipError_t = hipMemcpy(matrixA->d_rows, matrixA->h_rows, tamA*sizeof(float), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
	printf("hipMemcpy - A (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    hipError_t = hipMemcpy(matrixB->d_rows, matrixB->h_rows, tamB*sizeof(float), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
	printf("hipMemcpy - B (host to device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

   blockSize = THREADS_PER_BLOCK;
   numBlocks = (tamC + blockSize - 1) / blockSize;
   kernel_matrix_matrix_mult<<<numBlocks, blockSize>>>(*matrixA, *matrixB, *matrixC);

    hipDeviceSynchronize();

    hipError_t = hipMemcpy(matrixC->h_rows, matrixC->d_rows, tamC*sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy - C (device to host): returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 1;
    }

  return 1;
}