#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

#include "matrix_lib.h"
#include "timer.h"

Matrix *newMatrix (long int height, long int width){

    long int tam = height * width;
    hipError_t hipError_t;

    Matrix *matrix = (Matrix*) malloc(sizeof(Matrix));
    if(matrix == NULL){
        printf("(Error) Erro de memoria insuficiente\n");
        return NULL;
    }

    printf("(debug) Alocando host!\n");

    // HOST
    matrix->h_rows = (float *)aligned_alloc(32,2*tam*sizeof(float));
    if(matrix->h_rows == NULL){
        printf("(Error) Erro de memoria insuficiente\n");
        free(matrix);
        return NULL;
    }

    printf("(debug) Alocando device!\n");

    // DEVICE
    hipError_t = hipMalloc(&matrix->d_rows, tam*sizeof(float));

    if(hipError_t != hipSuccess){
        printf("(error) hipMalloc (device): returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        free(matrix);
        return NULL;
    }

    printf("(debug) Tudo alocado!\n");

    matrix->height = height;
    matrix->width = width;

    return matrix;
}

void initializeMatrix(Matrix *matrix) {
    long int tam = matrix->height * matrix->width;

    for(int i = 0 ; i < tam ; i++){
        matrix->h_rows[i] = 0;
    }
}

bool isValidDimension (long int height, long int width){
    if(height == width){
        return true;
    }
    return false;
}

void delMatrix(Matrix *matrix) {
    if(matrix != NULL) {
        // HOST
        printf("(debug) Liberando host\n");
        free(matrix->h_rows);
        // DEVICE
        printf("(debug) Liberando device\n");
        hipFree(matrix->d_rows);
        printf("(debug) Liberando matrix\n");
        free(matrix);
    }
}

void copyFloatArray (float *arr1, float *arr2, unsigned long int tam) {
    for(unsigned long int i = 0 ; i < tam ; i++) {
        arr1[i] = arr2[i];
    }
}

int main (int argc, char **argv) {
    // Lê parâmetros de execução =========================================================================
    // valor escalar
    float valorEscalar = (float)atoi(argv[1]);
    // matriz A
    long int dimMatrixA_height = atoi(argv[2]);
    long int dimMatrixA_width = atoi(argv[3]);
    // matriz Bs
    long int dimMatrixB_height = atoi(argv[4]);
    long int dimMatrixB_width = atoi(argv[5]);
    // número de threads por bloco a serem disparadas
    long int num_threads = atoi(argv[6]);
    // número máximo de blocos por GRID a serem usados
    long int max_block = atoi(argv[7]);
    // quantidade máxima de memória que pode ser alocada na GPGPU
    long int max_mem_gpu = atoi(argv[8]);
    // arquivo de floats
    char *arqFloats1 = argv[9];
    char *arqFloats2 = argv[10];
    // arquivo de resultados
    char *arqResult1 = argv[11];
    char *arqResult2 = argv[12];

    // Timer
    struct timeval start, stop, overall_t1, overall_t2;
    gettimeofday(&overall_t1, NULL);

    // Variáveis =======================================================================================
    unsigned long int tam, i;

    Matrix *matrixA;
    Matrix *matrixB;
    Matrix *matrixC;

    if(!isValidDimension(dimMatrixA_height,dimMatrixB_height)) {
        printf("(Error) Dimensões não são válidas\n");
        return 0;
    }

    matrixA = newMatrix(dimMatrixA_height, dimMatrixA_width);
    matrixB = newMatrix(dimMatrixB_height, dimMatrixB_width);
    matrixC = newMatrix(dimMatrixA_height, dimMatrixB_width);   // Dimensão de C := height de A e width de B

    tam = matrixC->height * matrixC->width;

    printf("(debug) Inicializando Matriz C!\n");
    initializeMatrix(matrixC);

    FILE *file_pointer;

    // Le arquivos =======================================================================================

    printf("(debug) Populando Matriz A!\n");

    // Arquivo 1
    file_pointer = fopen(arqFloats1,"rb");
    if (file_pointer == NULL) {
        printf("(Error) Erro ao tentar abrir o arquivo!\n");
        return 0;
    }
    // Popula ->rows
    fread(matrixA->h_rows, sizeof(matrixA->h_rows), matrixA->height*matrixA->width, file_pointer);
    fclose(file_pointer);

    printf("(debug) Populando Matriz B!\n");

    // Arquivo 2
    file_pointer = fopen(arqFloats2, "rb");
    if (file_pointer == NULL) {
        printf("(Error) Erro ao tentar abrir o arquivo!\n");
        return 0;
    }
    // Popula ->rows
    fread(matrixB->h_rows, sizeof(matrixB->h_rows), matrixB->height*matrixB->width, file_pointer);
    fclose(file_pointer);

    // Printa Matrizes ========================================================================================
    printf("--------Matriz A--------\n");
    for(unsigned long int i = 0; i < tam; i++){
        if(i > 256){
            printf(" -- A matriz passou do limite de 256 -- ");
            break;
        }
        printf("%.1f ", matrixA->h_rows[i]);
    }
    printf("\n");

    printf("--------Matriz B--------\n");
    for(unsigned long int i = 0; i < tam; i++){
        if(i > 256){
            printf(" -- A matriz passou do limite de 256 -- ");
            break;
        }
        printf("%.1f ", matrixB->h_rows[i]);
    }
    printf("\n");

    printf("--------Matriz C--------\n");
    for(unsigned long int i = 0; i < tam; i++){
        if(i > 256){
            printf(" -- A matriz passou do limite de 256 -- ");
            break;
        }
        printf("%.1f ", matrixC->h_rows[i]);
    }
    printf("\n");

    // Multiplicação Escalar ==================================================================================
    
    gettimeofday(&start, NULL);

    if(scalar_matrix_mult(valorEscalar, matrixA) == 0) {
        printf("(Error) Erro na multiplicação escalar\n");
    }

    gettimeofday(&stop, NULL);

    printf("Writing first result: %s\n", arqResult1);
    printf("Executing scalar_matrix_mult(%.1f, matrixA)\n", valorEscalar);

    // Show init exec time
    //printf("%f ms\n", timedifference_msec(start, stop));

    // printa a matriz
    printf("--------Matriz A--------\n");
    for(unsigned long int i = 0; i < matrixA->width * matrixA->height; i++){
        if(i > 256){
            printf(" -- A matriz passou do limite de 256 -- ");
            break;
        }
        printf("%.1f ", matrixA->h_rows[i]);
    }
    printf("\n");

    printf("(debug) Escrevendo no arquivo Result1\n");

    // Escreve arquivo binário Result1
    file_pointer = fopen(arqResult1, "wb");
    if (file_pointer == NULL) {
        printf("(Error) Erro ao tentar criar o arquivo!\n");
        return 0;
    }
    float* arrayAux1;
    arrayAux1 = (float*) malloc(dimMatrixA_height*dimMatrixA_width*sizeof(float)); // Array auxiliar para o fwrite();
    for(i = 0 ; i < matrixA->width * matrixA->height ; i++){
        arrayAux1[i] = matrixA->h_rows[i];
    }
    fwrite(arrayAux1, sizeof(arrayAux1), 1024 , file_pointer);
    fclose(file_pointer);

    // Multiplicação de Matrizes ======================================================================

    gettimeofday(&start, NULL);

    if(matrix_matrix_mult(matrixA, matrixB, matrixC) == 0) {
        printf("(Error) Erro na multiplicação de matrizes");
    }
    gettimeofday(&stop, NULL);
    printf("Writing second result: %s\n", arqResult2);
    printf("Executing matrix_matrix_mult(matrixA, matrixB, matrixC)\n");

    // Show init exec time
    //printf("%f ms\n", timedifference_msec(start, stop));

    // printa a matriz
    printf("--------Matriz C--------\n");
    for(unsigned long int i = 0; i < matrixC->width * matrixC->height; i++){
        if(i > 256){
            printf(" -- A matriz passou do limite de 256 -- ");
            break;
        }
        printf("%.1f ", matrixC->h_rows[i]);
    }
    printf("\n");
    // Escreve arquivo binario Result2
    file_pointer = fopen(arqResult2, "wb");
    if (file_pointer == NULL) {
        printf("(Error) Erro ao tentar criar o arquivo!\n");
        return 0;
    }
    float* arrayAux2;
    arrayAux2 = (float*) malloc(matrixC->height*matrixC->width*sizeof(float)); // Array auxiliar para o fwrite();
    for(i = 0 ; i < tam ; i++){
        arrayAux2[i] = matrixC->h_rows[i];
    }
    fwrite(arrayAux2, sizeof(arrayAux2), 1024 , file_pointer);
    fclose(file_pointer);
    // Le arquivo binario
    file_pointer = fopen(arqResult2, "rb");
    if (file_pointer == NULL) {
        printf("(Error) Erro ao tentar criar o arquivo!\n");
        return 0;
    }
    
    // TESTE DO ARQUIVO BINÁRIO
    //float matrixTest[tam]; // Array auxiliar para o fread();
    //fread(matrixTest, sizeof(matrixTest), 1, file_pointer);
    //fclose(file_pointer);
    //for(i = 0 ; i < tam ; i++) {
    //    printf("%f ", matrixTest[i]);
    //}
    //printf("\n");


    gettimeofday(&overall_t2, NULL);

    // Show elapsed overall time
    //printf("Overall time: %f ms\n", timedifference_msec(overall_t1, overall_t2));
    
    printf("(debug) Liberando memória\n");
    delMatrix(matrixA);
    delMatrix(matrixB);
    delMatrix(matrixC);
    printf("(debug) Fim\n");

    return 1;
}